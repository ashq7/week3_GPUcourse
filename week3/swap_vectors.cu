
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE = 10;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_swap(float *A, float *B, float *C, int v_size) {

    //FIXME:
    // Express the vector index in terms of threads and blocks
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Swap the vector elements - make sure you are not out of range
    if (idx < v_size){
       C[idx] = A[idx];
       A[idx] = B[idx];
       B[idx] = C[idx];
    }
}


int main() {


    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
        //question: why are the random numbers normally the same?
    }

    // Print initial elements to check swapping against later
    //question: does it matter if I print from device or host?
    printf("Matrix A: ");
    for (int i = 0; i < DSIZE; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    printf("Matrix B: ");
    for (int i = 0; i < DSIZE; i++) {
        printf("%f ", h_B[i]);
    }
    printf("\n");

    // Allocate memory for host and device pointers 
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*sizeof(float));

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    vector_swap<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);
    
    // Copy back to host 
    hipMemcpy(h_C, d_C, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successful
    printf("Matrix A post-swap: ");
    for (int i = 0; i < DSIZE; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    printf("Matrix B post-swap: ");
    for (int i = 0; i < DSIZE; i++) {
        printf("%f ", h_B[i]);
    }
    printf("\n");

    // Free the memory 
    //question: should host ones be freed?
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //question: do I need:
    hipDeviceSynchronize();
    return 0;
}
