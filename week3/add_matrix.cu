
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE_X = 4;
const int DSIZE_Y = 4;

__global__ void add_matrix(float* A, float* B, float* C, int N, int M)
{
    //FIXME:
    // Express in terms of threads and blocks
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;
    // Add the two matrices - make sure you are not out of range
    //question: why is there no "for" loop?
    if (idx <  N && idy < M ){
        C[idx * N + idy] =  A[idx * N + idy]+B[idx * N + idy];
    }
}

int main()
{

    // Create and allocate memory for host and device pointers 
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE_X * DSIZE_Y];
    h_B = new float[DSIZE_X * DSIZE_Y];
    h_C = new float[DSIZE_X * DSIZE_Y];

    hipMalloc(&d_A, DSIZE_X*DSIZE_Y*sizeof(float));
    hipMalloc(&d_B, DSIZE_X*DSIZE_Y*sizeof(float));
    hipMalloc(&d_C, DSIZE_X*DSIZE_Y*sizeof(float));

    // Fill in the matrices
    // FIXME
    for (int i = 0; i < DSIZE_X; i++) {
        for (int j = 0; j < DSIZE_Y; j++) {
            //FIXME
            h_A[i * DSIZE_X +j] = rand()/(float)RAND_MAX;
            h_B[i * DSIZE_X +j] = rand()/(float)RAND_MAX;
            h_C[i * DSIZE_X +j] = 0;
        }
    }

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    // dim3 is a built in CUDA type that allows you to define the block 
    // size and grid size in more than 1 dimentions
    // Syntax : dim3(Nx,Ny,Nz)
    int blockSize_x=1;
    int blockSize_y=1;
    dim3 blockSize(blockSize_x,blockSize_y); 
    dim3 gridSize(DSIZE_X/blockSize_x,DSIZE_Y/blockSize_y); //can I access blockSize entries?
    
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE_X, DSIZE_Y);

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make the addition was succesfull
    printf("Matrix A: ");
    for (int i = 0; i < DSIZE_X*DSIZE_Y; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    printf("Matrix B: ");
    for (int i = 0; i < DSIZE_X*DSIZE_Y; i++) {
        printf("%f ", h_B[i]);
    }
    printf("\n");

    printf("Matrix A + Matrix B: ");
    for (int i = 0; i < DSIZE_X*DSIZE_Y; i++) {
        printf("%f ", h_C[i]);
    }
    printf("\n");

    // Free the memory     
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}